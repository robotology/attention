#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <iCub/cudaVision/cudaVision.h>

////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[MAX_KERNEL_NUM][MAX_KERNEL_LENGTH];


extern "C" void setConvolutionKernel(float *h_Kernel, int k_Index, size_t k_Size)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel[k_Index]), h_Kernel, k_Size*sizeof(float));
}


#define   ADD_BLOCKDIM_X       16
#define   ADD_BLOCKDIM_Y       16
__global__ void addF32Kernel(float *C, float *A, float *B, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if(offset<N) 
    {
        float a = A[offset];
        float b = B[offset];
        C[offset] = a + b; 
    }
}

extern "C" void addF32(
    float *d_Dst,
    float *d_Src1,
    float *d_Src2,
    int imageW,
    int imageH)
{
    assert( imageW >= ADD_BLOCKDIM_X );
    assert( imageH >= ADD_BLOCKDIM_Y );
    dim3 dimBlock( ADD_BLOCKDIM_X, ADD_BLOCKDIM_Y );
    dim3 dimGrid( ceil(float(imageW)/float(ADD_BLOCKDIM_X)), 
                  ceil(float(imageH)/float(ADD_BLOCKDIM_Y)) );                  
    addF32Kernel<<<dimGrid, dimBlock>>>(d_Dst, d_Src1, d_Src2, imageW*imageH);
}


// Row convolution filter
#define   ROWS_BLOCKDIM_X       32//16
#define   ROWS_BLOCKDIM_Y       4
#define   ROWS_RESULT_STEPS     5//8
#define   ROWS_HALO_STEPS       1

__global__ void convRowsF32SepKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int k_index,
    int k_radius)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
    #pragma unroll
    for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];

    //Load left halo
    #pragma unroll
    for(int i = 0; i < ROWS_HALO_STEPS; i++)
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X ) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

    //Load right halo
    #pragma unroll
    for(int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

    //Compute and store results
    __syncthreads();
    #pragma unroll
    for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++){
        float sum = 0;

        #pragma unroll
        for(int j = -k_radius; j <= k_radius; j++)
            sum += c_Kernel[k_index][k_radius - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

extern "C" void convRowsF32Sep(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int k_Index,
    int k_Size)
{
    int k_radius = ((k_Size % 2) == 0) ? k_Size/2 : (k_Size-1)/2;
    assert( ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= k_radius );
    assert( imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0 );
    assert( imageH % ROWS_BLOCKDIM_Y == 0 );

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convRowsF32SepKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW,
        k_Index,
        k_radius);
    //cutilCheckMsg("convolutionRowsKernel() execution failed\n");
}




// Column convolution filter
#define   COLUMNS_BLOCKDIM_X        16
#define   COLUMNS_BLOCKDIM_Y        16 //8
#define   COLUMNS_RESULT_STEPS      5 //8
#define   COLUMNS_HALO_STEPS        1

__global__ void convColsF32SepKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int k_index,
    int k_radius)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
    #pragma unroll
    for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];

    //Upper halo
    #pragma unroll
    for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

    //Lower halo
    #pragma unroll
    for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

    //Compute and store results
    __syncthreads();
    #pragma unroll
    for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++){
        float sum = 0;
        #pragma unroll
        for(int j = -k_radius; j <= k_radius; j++)
            sum += c_Kernel[k_index][k_radius - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

extern "C" void convColsF32Sep(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int k_Index,
    int k_Size)
{

    int k_radius = ((k_Size % 2) == 0) ? k_Size/2 : (k_Size-1)/2;
    assert( COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= k_radius );
    assert( imageW % COLUMNS_BLOCKDIM_X == 0 );
    assert( imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0 );

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convColsF32SepKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW,
        k_Index,
        k_radius
    );
    //cutilCheckMsg("convolutionColumnsKernel() execution failed\n");
}


extern "C" void convF32Sep(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int k_Index,
    int k_Size,
    float* d_Buffer)
{

    if(d_Buffer == NULL)
    {
        HANDLE_ERROR( hipMalloc((void **)&d_Buffer, imageW*imageH*sizeof(float)) );   
        convRowsF32Sep(d_Buffer, d_Src, imageW, imageH, k_Index, k_Size);
        convColsF32Sep(d_Dst, d_Buffer, imageW, imageH, k_Index, k_Size);
        HANDLE_ERROR( hipFree(d_Buffer ) );
    }
    else
    {
        convRowsF32Sep(d_Buffer, d_Src, imageW, imageH, k_Index, k_Size);
        convColsF32Sep(d_Dst, d_Buffer, imageW, imageH, k_Index, k_Size);
    }
}


extern "C" void convF32SepAdd(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int k_Index,
    int k_Size,
    float* d_Buffer)
{

    if(d_Buffer == NULL)
    {
        HANDLE_ERROR( hipMalloc((void **)&d_Buffer, imageW*imageH*sizeof(float)) );   
        convRowsF32Sep(d_Buffer, d_Src, imageW, imageH, k_Index, k_Size);
        HANDLE_ERROR( hipDeviceSynchronize() );
        convColsF32Sep(d_Dst, d_Src, imageW, imageH, k_Index, k_Size);
        addF32(d_Dst, d_Dst, d_Buffer, imageW, imageH); 
        HANDLE_ERROR( hipDeviceSynchronize() );
        HANDLE_ERROR( hipFree(d_Buffer ) );
    }
    else
    {
        convRowsF32Sep(d_Buffer, d_Src, imageW, imageH, k_Index, k_Size);
        HANDLE_ERROR( hipDeviceSynchronize() );
        convColsF32Sep(d_Dst, d_Src, imageW, imageH, k_Index, k_Size);
        HANDLE_ERROR( hipDeviceSynchronize() );
        addF32(d_Dst, d_Dst, d_Buffer, imageW, imageH);        
        HANDLE_ERROR( hipDeviceSynchronize() );
    }
}



/*
////////////////////////////////////////////////////////////////////////////////
// Texture convolution
////////////////////////////////////////////////////////////////////////////////
//Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

//Use unrolled innermost convolution loop
//#define UNROLL_INNER 1

//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

texture<float, 2, hipReadModeElementType> texSrc;

extern "C" void setInputArray(hipArray *a_Src){
}

extern "C" void detachInputArray(void){
}
*/

/*
// Loop unrolling templates, needed for best performance
template<int i> __device__ float convolutionRow(float x, float y){
    return 
        tex2D(texSrc, x + (float)(KERNEL_RADIUS - i), y) * c_Kernel[i]
        + convolutionRow<i - 1>(x, y);
}

template<> __device__ float convolutionRow<-1>(float x, float y){
    return 0;
}

template<int i> __device__ float convolutionColumn(float x, float y){
    return 
        tex2D(texSrc, x, y + (float)(KERNEL_RADIUS - i)) * c_Kernel[i]
        + convolutionColumn<i - 1>(x, y);
}

template<> __device__ float convolutionColumn<-1>(float x, float y){
    return 0;
}
*/

/*
// Row convolution filter
__global__ void convRowsF32TexKernel(
    float *d_Dst,
    int imageW,
    int imageH,
    int k_radius)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix >= imageW || iy >= imageH)
        return;

    float sum = 0;

    //#if(UNROLL_INNER)
    //    sum = convolutionRow<2 * k_radius>(x, y);
    //#else
        for(int k = -k_radius; k <= k_radius; k++)
        sum += tex2D(texSrc, x + (float)k, y) * c_Kernel[k_radius - k];
    //#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}


extern "C" void convRowsF32Tex(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH,
    int k_radius)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    HANDLE_ERROR( hipBindTextureToArray(texSrc, a_Src) );
    convRowsF32TexKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH,
        k_radius
    );
    HANDLE_ERROR( hipUnbindTexture(texSrc) );
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convColsF32TexKernel(
    float *d_Dst,
    int imageW,
    int imageH,
    int k_radius)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix >= imageW || iy >= imageH)
        return;

    float sum = 0;

    //#if(UNROLL_INNER)
    //    sum = convolutionColumn<2 * k_radius>(x, y);
    //#else
        for(int k = -k_radius; k <= k_radius; k++)
            sum += tex2D(texSrc, x, y + (float)k) * c_Kernel[k_radius - k];
    //#endif

     d_Dst[IMAD(iy, imageW, ix)] = sum;
}


extern "C" void convColsF32Tex(
    float *d_Dst,
    hipArray *a_Src,
    int imageW,
    int imageH,
    int k_radius)
{
    dim3 threads(16, 12);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    HANDLE_ERROR(hipBindTextureToArray(texSrc, a_Src) );
    convColsF32TexKernel<<<blocks, threads>>>(
        d_Dst,
        imageW,
        imageH,
        k_radius);
    HANDLE_ERROR( hipUnbindTexture(texSrc) );
}
*/

